#include "hip/hip_runtime.h"
/*
 * c2g.cu
 *
 *  Created on: 18/07/2011
 *      Author: wendell & agnus
 */

// includes, system
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <ctime>
//#include <cmath>
#include <iostream>
#include "amy_colors.h"

// includes, kernels
#include <c2g.cuh>

using namespace std;

__host__ __device__ int iDivUp(int a, int b)
{
	return a / b + (a % b != 0);
}

__constant__ float cAlpha, cSTheta, cCTheta;
__constant__ int cN;

__global__ void cuDColor2Gray_kernel(amy_lab *dataIn, float *dataOut)
{

	const int id = (blockDim.x * blockIdx.x) + threadIdx.x;

	if (id >= cN)
		return;

	float sum = 0;
	int nSigma = 0;

#pragma unroll
	for (int j = 0; j < cN; ++j)
	{

		float Lij = dataIn[id].l - dataIn[j].l;

		//Calculo de Lj + deltaij
		if (fabs(Lij) <= cAlpha)
		{
			float dij;

			float Cij[] =
			{ (dataIn[id].a - dataIn[j].a), (dataIn[id].b - dataIn[j].b) };

			float crunch = CRUNCH(cAlpha, L2NORM(Cij[0], Cij[1]));

			if (fabs(Lij) > crunch)
			{
				dij = dataIn[id].l;
			}
			else
			{
				float sign = DOT(Cij[0], Cij[1], cCTheta, cSTheta);
				dij = sign > 0 ? crunch : -crunch;
				dij += dataIn[j].l;
			}
			sum += dij;
			++nSigma;
		}
	}
	dataOut[id] = ((((cN) - nSigma) * dataIn[id].l + sum) / (cN));
	//	dataOut[id] = sum;
}

__global__ void cuColor2Gray_kernel(amy_lab *dataIn, float *dataOut)
{
	const int id = (blockDim.x * blockIdx.x) + threadIdx.x;

	__shared__
	amy_lab shImg[SH_1D];

	float sum = 0;
	int nSigma = 0;

	register float Li = dataIn[id].l;
	register float ai = dataIn[id].a;
	register float bi = dataIn[id].b;
	register unsigned int pixelCount = 0;

	for (int i = 0; i < iDivUp(cN, SH_1D); ++i)
	{
		//shImg[threadIdx.x] = threadIdx.x + i * BLOCK_1D < w * h ? dataIn[threadIdx.x + i * BLOCK_1D] : zero;
		//		for (int k = 0; k < 4; ++k)
		//		{
		//			if (threadIdx.x + i * BLOCK_1D < cN)
		//			{
		//				shImg[threadIdx.x + k*BLOCK_1D] = dataIn[threadIdx.x + i * BLOCK_1D];
		//			}
		//		}
		if (threadIdx.x + i * SH_1D < cN)
		{
			shImg[threadIdx.x] = dataIn[threadIdx.x + i * SH_1D];
			shImg[threadIdx.x + BLOCK_1D] = dataIn[threadIdx.x + i * SH_1D + BLOCK_1D];
			shImg[threadIdx.x + BLOCK_1D*2] = dataIn[threadIdx.x + i * SH_1D + BLOCK_1D*2];
			shImg[threadIdx.x + BLOCK_1D*3] = dataIn[threadIdx.x + i * SH_1D + BLOCK_1D*3];

		__syncthreads();
		for (int j = 0; j < SH_1D && pixelCount < cN; ++j)
		{
			float Lij = Li - shImg[j].l;

			//Calculo de Lj + deltaij
			if (abs(Lij) <= cAlpha)
			{
				float dij;

				float Cij[] =
				{ (ai - shImg[j].a), (bi - shImg[j].b) };

				float crunch = CRUNCH(cAlpha, L2NORM(Cij[0], Cij[1]));

				if (abs(Lij) > crunch)
				{
					dij = Li;
				}
				else
				{
					float sign = DOT(Cij[0], Cij[1], cCTheta, cSTheta);
					dij = sign > 0 ? crunch : -crunch;
					dij += shImg[j].l;
				}
				sum += dij;
				++nSigma;
			}
			++pixelCount;
		}
		__syncthreads();
		}
	}
	if(id >= cN) return;
	dataOut[id] = ((((cN) - nSigma) * Li + sum) / (cN));
}

void cuColor2Gray(amy_lab *dataIn, float *dataOut, int n, float alpha,
		float ctheta, float stheta)
{

	int grid(iDivUp(n, BLOCK_1D));
	int block(BLOCK_1D);

	//	cout << "G: " << grid << " B: " << block << endl;

	//Ponteiros para o device
	amy_lab *devDataIn;
	float *devDataOut;

	size_t sizeAmy = n * sizeof(amy_lab);
	size_t sizeF = n * sizeof(float);

	hipMalloc((void **) &devDataIn, sizeAmy);
	hipMalloc((void **) &devDataOut, sizeF);

	hipMemcpy(devDataIn, dataIn, sizeAmy, hipMemcpyHostToDevice);
	hipMemcpy(devDataOut, dataOut, sizeF, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(cN), &n, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(cAlpha), &alpha, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(cSTheta), &stheta, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(cCTheta), &ctheta, sizeof(float));

	//	hipMemset(devDataOut, 1, sizeF);

	//	cout << "dataIn:" << endl;
	//	for (int i = 0; i < width * height; ++i)
	//	{
	//		cout << dataIn[i].l << " ";
	//		if ((i + 1) % width == 0)
	//			cout << endl;
	//	}

	cuColor2Gray_kernel<<<grid, block>>>(devDataIn, devDataOut);

	hipMemcpy(dataOut, devDataOut, sizeF, hipMemcpyDeviceToHost);

	//	cout << "dataOut:" << endl;
	//	for (int i = 0; i < n; ++i)
	//	{
	//		cout << dataOut[i] << " ";
	//		if ((i + 1) % 100 == 0)
	//			cout << endl;
	//	}

	//	cout << "Diff:" << endl;
	//	for (int i = 0; i < width*height; ++i) {
	//			cout << dataOut[i] - dataIn[i].l << " ";
	//			if((i+1)% width == 0) cout << endl;
	//		}

	hipFree(devDataIn);
	hipFree(devDataOut);

}

